#include "hip/hip_runtime.h"
// ref:
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSPARSE/spsv_csr/spsv_csr_example.c

#include "common.h"
#include "mmio.h"
#include "read_mtx.h"
#include "tranpose.h"
#include "YYSpTRSV.h"

#include "ArrayUtils.hpp"

using namespace uni;

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            while (1)                                                  \
                ;                                                      \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            while (1)                                                      \
                ;                                                          \
        }                                                                  \
    }

using cusp_int = int;
#define my_CUSPARSE_INDEX HIPSPARSE_INDEX_32I

#define MAX_DOF_TEST 1

struct benchmark_record
{
    double total_time = 0;
    long flops = 0;
    long bytes = 0;
    long count = 0;
};

benchmark_record benchmark_record_map_lower[MAX_DOF_TEST];
benchmark_record benchmark_record_map_upper[MAX_DOF_TEST];

template <int Dim = 3, int stencil_type = 0, int stencil_width>
void RunBenchmarkLowerWithCusparse(cusp_int M, cusp_int N, cusp_int P, int Dof)
{
    std::string dof_str = std::to_string(Dof);
    // log::FunctionBegin("RunBenchmark_Dof_" + dof_str + "_WithCusparse");
    // cusp_int M = json[dof_str]["M"].get<cusp_int>();
    // cusp_int N = json[dof_str]["N"].get<cusp_int>();
    // cusp_int P = json[dof_str]["P"].get<cusp_int>();

    // for ilu(1)
    // StencilPattern<Dim> stencil_pattern(
    //     ((stencil_type == 0 || stencil_type == 2) ? STENCIL_STAR : STENCIL_BOX),
    //     stencil_width);
    // StencilPattern<Dim> lower_fill_pattern =
    //     stencil_pattern.GetFillInPattern(1).GetLowerWithDiagPattern();

    std::vector<std::array<cusp_int, Dim>> stencil_points;
    if constexpr (stencil_type == 0)
    {
        for (int d = Dim - 1; d >= 0; d--)
        {
            for (int j = stencil_width; j > 0; j--)
            {
                std::array<cusp_int, Dim> pt = {0, 0, 0};
                pt[d] = -j;
                stencil_points.push_back(pt);
            }
        }
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, 0});
    }
    // else if constexpr (stencil_type == 1) {
    //     NestedLoop(
    //         constant_array<cusp_int, Dim>(-stencil_width),
    //         constant_array<cusp_int, Dim>(2 * stencil_width + 1), [&](auto pt) {
    //             IndexInt cnt = CartToFlat(
    //                 pt + stencil_width,
    //                 constant_array<IndexInt, Dim>(2 * stencil_width + 1));
    //             if (cnt < (myPow(2 * stencil_width + 1, Dim) / 2)) {
    //                 stencil_points.push_back(pt);
    //             }
    //         });
    //     stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, 0});
    // } else {
    //     for (int cnt = 0; cnt < lower_fill_pattern.length; cnt++) {
    //         stencil_points.push_back(lower_fill_pattern.points(cnt));
    //     }
    // }

    // Host problem definition
    cusp_int A_num_rows = M * N * P * Dof;
    cusp_int A_num_cols = M * N * P * Dof;
    cusp_int A_nnz = 0;
    std::vector<cusp_int> hA_csrOffsets;
    std::vector<cusp_int> hA_columns;
    std::vector<double> hA_values;
    std::vector<double> hX;
    std::vector<double> hY;
    std::vector<double> hY_result;
    double alpha = 1.0f;
    // 注意这里求解的是A* Y = X, 所以这里的Y是输出, X是输入

    // set A & hX
    NestedLoop(
        std::array<cusp_int, Dim>{}, std::array<cusp_int, Dim>{M, N, P},
        [&](auto loc)
        {
            for (int d = 0; d < Dof; d++)
            {
                hA_csrOffsets.push_back(A_nnz);
                cusp_int cnt = 0;
                for (auto pt : stencil_points)
                {
                    if (in_range(loc + pt, std::array<cusp_int, Dim>{},
                                 std::array<cusp_int, Dim>{M, N, P} - 1))
                    {
                        for (int k = 0; k < Dof; k++)
                        {
                            hA_columns.push_back(
                                CartToFlat(loc + pt,
                                           std::array<cusp_int, Dim>{M, N, P}) *
                                    Dof +
                                k);
                            hA_values.push_back(1.);
                            A_nnz++;
                            cnt++;
                        }
                    }
                }
                hX.push_back(cnt);
            }
        });
    hA_csrOffsets.push_back(A_nnz);

    std::cout << "A_nnz = " << A_nnz << std::endl;

    // set hY
    hY.resize(A_num_cols);
    hY_result.resize(A_num_rows);
    for (cusp_int i = 0; i < A_num_cols; i++)
        hY_result[i] = 1.0;

    //--------------------------------------------------------------------------

    /* The border between thread-level and warp-level algorithms, according to
     * the number of non-zero elements in each row of the matrix L*/
    int border = 10;

    /* !!!!!! start computing SpTRSV !!!!!!!! */
    double solve_time, gflops, bandwith, pre_time, warp_occupy, element_occupy;
    int success = YYSpTRSV_csr(
        A_num_rows, A_num_cols, A_nnz, hA_csrOffsets.data(), hA_columns.data(),
        hA_values.data(), hX.data(), hY.data(), border, &solve_time, &gflops,
        &bandwith, &pre_time, &warp_occupy, &element_occupy);

    long readBytes = (sizeof(cusp_int) + sizeof(double)) * A_nnz +
                     sizeof(cusp_int) * A_num_rows +
                     sizeof(double) * A_num_cols;
    long writeBytes = sizeof(double) * A_num_rows;

    // double timing = 0;
    // log::FunctionBegin("Timing");
    // timing
    // for (int i = 0; i < 10; i++) {
    //     log::FunctionBegin("hipsparseSpMV");
    //     double time_0 = MPI_Wtime();
    //     CHECK_CUSPARSE(hipsparseSpSV_solve(
    //         handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX,
    //         vecY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr))
    //     CHECK_CUDA(hipDeviceSynchronize())
    //     double time_1 = MPI_Wtime();
    //     log::FunctionEnd(2 * A_nnz, readBytes, writeBytes);
    //     timing += time_1 - time_0;
    // }
    // log::FunctionEnd(0, 0, 0);
    benchmark_record_map_lower[Dof - 1] = {solve_time, 2 * A_nnz,
                                           (readBytes + writeBytes), 1};
    std::cout
        << "LowerTime: " << solve_time << ", Gflops: " << gflops << std::endl;

    //--------------------------------------------------------------------------
    // device result check

    int correct = 1;
    for (cusp_int i = 0; i < A_num_rows; i++)
    {
        if (hY[i] !=
            hY_result[i])
        {                // direct doubleing point comparison is not
            correct = 0; // reliable
            // break;
            std::cout << "i = " << i << ", hY[i] = " << hY[i]
                      << ", hY_result[i] = " << hY_result[i] << std::endl;
        }
    }
    if (correct)
        printf("spmv_csr_example test PASSED\n");
    else
        printf("spmv_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    // log::FunctionEnd(0, 0, 0);
}

int main(int argc, char **argv)
{
    // Json json =
    //     LoadJsonFromFile("example/structure-benchmark/matsolve-yysptrsv.json");
    // std::string platform = json["platform"];
    // std::string remark = json["remark"];
    // std::string problems[] = {"stencilstar", "stencilbox", "stencilstarfill1"};
    // bool if_output = json["output"];
    // MPI_Init(&argc, &argv);
    // int rank, size;
    // MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    // MPI_Comm_size(MPI_COMM_WORLD, &size);
    // loop<int, 1>([&](auto i) {
    // loop<int, (1)>([&](auto stencil_width_0) {
    // constexpr int stencil_width = stencil_width_0 + 1;
    // std::string problem = problems[i];
    // if (if_output) {
    //     Logger::InitSingleOutput(
    //         rank, std::string{"results/matsolve-yysptrsv-"} + problem +
    //                   "-stencilwidth" + std::to_string(stencil_width) +
    //                   "-" + platform + "-" + remark + ".out");
    // } else {
    //     Logger::Init();
    // }
    // loop<int, MAX_DOF_TEST>([&](auto dof) {
    // Logger::output(ConsoleAndRank)
    //     << problem << ", width=" << stencil_width
    //     << ", dof=" << dof + 1 << std::endl;
    // log::FunctionBegin(problem + "_lower");
    RunBenchmarkLowerWithCusparse<3, 0, 1>(
        192, 192, 192, 1);
    // log::FunctionEnd(0, 0, 0);
    // Logger::output(ConsoleAndRank) << "Lower:";
    // double total_time = benchmark_record_map_lower[dof].total_time +
    //                     benchmark_record_map_upper[dof].total_time;
    // double total_flops_time =
    //     static_cast<double>(benchmark_record_map_lower[dof].flops +
    //                         benchmark_record_map_upper[dof].flops) /
    //     total_time;
    // double total_bytes_time =
    //     static_cast<double>(benchmark_record_map_lower[dof].bytes +
    //                         benchmark_record_map_upper[dof].bytes) /
    //     total_time;

    // Logger::output(ConsoleAndRank)
    //     << dof + 1 << "," << total_time << ","
    //     << total_flops_time * 1e-9 << "," << total_bytes_time * 1e-9
    //     << std::endl;
    // });
    // log::ReportResult(ConsoleAndRank);
    // log::ClearReportResult();
    // Logger::Finalize();
    // });
    // });
    // MPI_Finalize();
    return 0;
}
